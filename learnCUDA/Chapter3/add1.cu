
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 4.56;
const double c = 5.79;
void __global__ add(double *x, double *y, double *z, const int N);
void check(double *z, const int N);

int main(void)
{
    // Use CUDA Compiler: nvcc -arch=sm_75 add1.cu -o add1
    // Run exe file: ./add
    // Codes run on GPU
    
    const int N = 100000000;
    const int M = N * sizeof(double);

    double *h_x = (double*) malloc(M);
    double *h_y = (double*) malloc(M);
    double *h_z = (double*) malloc(M);

    for(int i=0;i<N;i++)
    {
        h_x[i] = a;
        h_y[i] = b;
    }

    double *d_x, *d_y, *d_z;
    hipMalloc((void **) &d_x, M);  // Alloc memory on device
    hipMalloc((void **) &d_y, M);  // force double** input &d_x to be void **
    hipMalloc(&d_z, M);  // without void ** the cudaMalloc can finish transformation as well

    // Transfer data from host to device
    hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice);

    // grid_size=781250, it can exceeds max gridDim.x if you use CUDA 8.0 or lower version
    const int block_size = 128;
    const int grid_size = int(N / block_size);

    // params in <<<>>> have to be appoined
    add<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    // Wrong transfer direction will cause wrong errors
    // cudaMemcpy(h_z, d_z, M, cudaMemcpyHostToDevice);
    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);
    check(h_z, N);

    // notice the cuda data must be freed by cudaFree() 
    // and host data must be freed by free()
    // free(d_x);  // otherwise it would raise "Segmentation fault (core dumped)"
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);

    free(h_x);
    free(h_y);
    free(h_z);

    return 0;
}

// The number of inputs has to be fixed
// non-pointer inputs are visible for all threads
// pointer inputs have to point to device memory
// It can 
void __global__ add(double *x, double *y, double *z, const int N)
{
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    z[n] = x[n] + y[n];
}


void check(double *z, const int N)
{
    bool has_error = false;

    for(int i = 0; i < N; i++)
        if(abs(z[i] - c) > EPSILON)
        {
            has_error = true;
            break;
        }
    printf("%s\n", has_error ? "Has Errors" : "No Errors");
}

