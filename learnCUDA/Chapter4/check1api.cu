#include "hip/hip_runtime.h"
#include "error.cuh"
#include <stdio.h>
#include <math.h>

const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 4.56;
const double c = 5.79;
void __global__ add(double *x, double *y, double *z, const int N);
void check(double *z, const int N);

int main(void)
{
    // Use CUDA Compiler: nvcc -arch=sm_75 check1api.cu -o check1api
    // Run exe file: ./check1api
    // Codes run on GPU
    
    const int N = 100000000;
    const int M = N * sizeof(double);

    double *h_x = (double*) malloc(M);
    double *h_y = (double*) malloc(M);
    double *h_z = (double*) malloc(M);

    for(int i=0;i<N;i++)
    {
        h_x[i] = a;
        h_y[i] = b;
    }

    double *d_x, *d_y, *d_z;
    CHECK(hipMalloc((void **) &d_x, M));  // Alloc memory on device
    CHECK(hipMalloc((void **) &d_y, M));  // force double** input &d_x to be void **
    CHECK(hipMalloc(&d_z, M));  // without void ** the hipMalloc can finish transformation as well

    // Transfer data from host to device
    CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice));

    // grid_size=781250, it can exceeds max gridDim.x if you use CUDA 8.0 or lower version
    const int block_size = 128;
    const int grid_size = int((N + block_size - 1) / block_size);

    // params in <<<>>> have to be appoined
    add<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    // Wrong transfer direction will cause wrong errors
    CHECK(hipMemcpy(h_z, d_z, M, hipMemcpyHostToDevice));
    // CHECK(hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost));
    check(h_z, N);

    // notice the cuda data must be freed by hipFree() 
    // and host data must be freed by free()
    // free(d_x);  // otherwise it would raise "Segmentation fault (core dumped)"
    CHECK(hipFree(d_x));
    CHECK(hipFree(d_y));
    CHECK(hipFree(d_z));

    free(h_x);
    free(h_y);
    free(h_z);

    return 0;
}

// The number of inputs has to be fixed
// non-pointer inputs are visible for all threads
// pointer inputs have to point to device memory
// It can 
void __global__ add(double *x, double *y, double *z, const int N)
{
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    z[n] = x[n] + y[n];
}


void check(double *z, const int N)
{
    bool has_error = false;

    for(int i = 0; i < N; i++)
        if(abs(z[i] - c) > EPSILON)
        {
            has_error = true;
            break;
        }
    printf("%s\n", has_error ? "Has Errors" : "No Errors");
}

