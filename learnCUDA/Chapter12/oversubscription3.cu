#include "error.cuh"
#include <stdio.h>
#include <stdint.h>

const int N = 96;

void cpu_touch(uint64_t *x, const size_t size)
{
    for(int idx = 0; idx < size; idx++)
        x[idx] = 0;
}

int main(void)
{
    // Use -O3 option for highest level optimization
    // Usage: nvcc -O3 -arch=sm_75 oversubscription3.cu -o oversubscription3.out
    // Run exe file: cuda-memcheck ./oversubscription3.out

    for (int n = 8; n <= N; n += 8)
    {
        const size_t memory_size = size_t(n) * 1024 * 1024 * 1024;
        const size_t data_size = memory_size / sizeof(uint64_t);
        uint64_t *x;
        CHECK(hipMallocManaged(&x, memory_size));
        cpu_touch(x, data_size);
        CHECK(hipGetLastError());
        CHECK(hipDeviceSynchronize());
        CHECK(hipFree(x));
        printf("Allocated %d GB unified memory with CPU touch.\n", n);
    }
    return 0;
}

