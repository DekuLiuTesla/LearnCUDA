#include "hip/hip_runtime.h"
#include "error.cuh"
#include <math.h>
#include <stdio.h>

#ifdef USE_DP
    typedef double real;
    const real EPSILON = 1.0e-15;
#else
    typedef float real;
    const real EPSILON = 1.0e-6f;
#endif

const int NUM_REPEATS = 10;
const real a = 1.23;
const real b = 2.34;
const real c = 3.57;
void __global__ add(real *x, real *y, real *z, const int N);
void check(real *z, const int N);

int main(void)
{
    // Use -O3 option for highest level optimization
    // In float precision: nvcc -O3 -arch=sm_75 add.cu -o add.out
    // In real precision: nvcc -O3 -arch=sm_75 -DUSE_DP add.cu -o add.out
    // Run exe file: cuda-memcheck ./add.out

    const int N = 100000000;
    const int M = sizeof(real) * N;
    real *x, *y, *z;
    CHECK(hipMallocManaged((void **)&x, M));
    CHECK(hipMallocManaged((void **)&y, M));
    CHECK(hipMallocManaged((void **)&z, M));

    for(int i=0;i<N;i++)
    {
        x[i] = a;
        y[i] = b;
    }

    // grid_size=781250, it can exceeds max gridDim.x if you use CUDA 8.0 or lower version
    const int block_size = 128;
    const int grid_size = int((N + block_size - 1) / block_size);

    float t_sum = 0;
    float t2_sum = 0;
    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        // make sure hipEventRecord run on GPU in WDDM mode
        hipEventQuery(start);

        add<<<grid_size, block_size>>>(x, y, z, N);

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / NUM_REPEATS;  // mean
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);  // std
    printf("Time = %g +- %g ms.\n", t_ave, t_err);

    check(z, N);

    CHECK(hipFree(x));
    CHECK(hipFree(y));
    CHECK(hipFree(z));
    
    return 0;
}

// The number of inputs has to be fixed
// non-pointer inputs are visible for all threads
// pointer inputs have to point to device memory
void __global__ add(real *x, real *y, real *z, const int N)
{
    // mainly dominated by memory access
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    z[n] = x[n] + y[n];
}


void check(real *z, const int N)
{
    bool has_error = false;

    for(int i = 0; i < N; i++)
        if(abs(z[i] - c) > EPSILON)
        {
            has_error = true;
            break;
        }
    printf("%s\n", has_error ? "Has Errors" : "No Errors");
}
