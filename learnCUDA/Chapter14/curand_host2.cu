#include "error.cuh"
#include <hiprand.h>
#include <stdlib.h>
#include <stdio.h>

void output_results(int N, double *g_x);

int main()
{
    // Use -O3 option for highest level optimization
    // Compilation: nvcc -O3 -arch=sm_75 -lcurand curand_host2.cu -o curand_host2.out
    // Run exe file: cuda-memcheck ./curand_host2.out

    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234);

    int N = 100000;
    double *g_x;
    hipMalloc(&g_x, N * sizeof(double));
    hiprandGenerateNormalDouble(gen, g_x, N, 0.0, 1.0);

    double *x = (double *)malloc(N * sizeof(double));
    hipMemcpy(x, g_x, N * sizeof(double), hipMemcpyDeviceToHost);
    output_results(N, x);
    
    hipFree(g_x);
    hiprandDestroyGenerator(gen);
    free(x);

    return 0;
}

void output_results(int N, double *x)
{
    FILE *fid = fopen("x2.txt", "w");
    for (int i = 0; i < N; i++)
    {
        fprintf(fid, "%g\n", x[i]);
    }
    fclose(fid);
}