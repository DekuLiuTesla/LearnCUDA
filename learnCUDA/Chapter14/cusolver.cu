#include "error.cuh"
#include <hipsolver.h>
#include <stdio.h>
#include <stdlib.h>

int main()
{
    // Use -O3 option for highest level optimization
    // Compilation: nvcc -O3 -arch=sm_75 -lcusolver cusolver.cu -o cusolver.out
    // Run exe file: cuda-memcheck ./cusolver.out

    int N = 2;
    int N2 = N * N;
    hipDoubleComplex *A_cpu = (hipDoubleComplex *)malloc(N2 * sizeof(hipDoubleComplex));
    A_cpu[0].x = 0;
    A_cpu[1].x = 0;
    A_cpu[2].x = 0;
    A_cpu[3].x = 0;
    A_cpu[0].y = 0; 
    A_cpu[1].y = 1;
    A_cpu[2].y = -1;
    A_cpu[3].y = 0;
    hipDoubleComplex *A_gpu;
    CHECK(hipMalloc((void **)&A_gpu, N2 * sizeof(hipDoubleComplex)));
    CHECK(hipMemcpy(A_gpu, A_cpu, N2 * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

    double *W_cpu = (double *)malloc(N * sizeof(double));
    double *W_gpu;
    CHECK(hipMalloc((void **)&W_gpu, N * sizeof(double)));

    hipsolverHandle_t handle = NULL;
    hipsolverDnCreate(&handle);
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    int lwork = 0;
    hipsolverDnZheevd_bufferSize(handle, jobz, uplo, N, A_gpu, N, W_gpu, &lwork);
    hipDoubleComplex *work;
    CHECK(hipMalloc((void **)&work, lwork * sizeof(hipDoubleComplex)));

    int *devInfo;
    CHECK(hipMalloc((void **)&devInfo, sizeof(int)));
    hipsolverDnZheevd(handle, jobz, uplo, N, A_gpu, N, W_gpu, work, lwork, devInfo);

    hipMemcpy(W_cpu, W_gpu, N * sizeof(double), hipMemcpyDeviceToHost);
    printf("Eigenvalues:\n");
    for (int i = 0; i < N; i++)
    {
        printf("%f\n", W_cpu[i]);
    }

    hipsolverDnDestroy(handle);

    free(A_cpu);
    free(W_cpu);

    CHECK(hipFree(A_gpu));
    CHECK(hipFree(W_gpu));
    CHECK(hipFree(work));
    CHECK(hipFree(devInfo));

    return 0;
}
}