#include "hip/hip_runtime.h"
#include "error.cuh"
#include <stdio.h>

#ifdef USE_DP
    typedef double real;
    const real EPSILON = 1.0e-15;
#else
    typedef float real;
    const real EPSILON = 1.0e-6f;
#endif

const int NUM_REPEATS = 10;
const int TILE_DIM = 32;  // can't directly use address or reference of it in kernel function

void timing(const real *d_A, real *d_B, const int N, const int task);
__global__ void copy(const real *A, real *B, const int N);
__global__ void transpose1(const real *A, real *B, const int N);
__global__ void transpose2(const real *A, real *B, const int N);
__global__ void transpose3(const real *A, real *B, const int N);
void print_matrix(const int N, const real *A);

int main(int argc, char **argv)
{
    // Use CUDA Compiler: nvcc -arch=sm_75 matrix.cu -o matrix.out
    // Run exe file: ./matrix.out 10000
    if (argc != 2)
    {
        printf("usage: %s N\n", argv[0]);
        exit(1);
    }
    const int N = atoi(argv[1]);

    const int N2 = N * N;
    const int M = sizeof(real) * N2;
    real *h_A = (real *) malloc(M);
    real *h_B = (real *) malloc(M);
    for (int n = 0; n < N2; ++n)
    {
        h_A[n] = n;
    }
    real *d_A, *d_B;
    CHECK(hipMalloc(&d_A, M));
    CHECK(hipMalloc(&d_B, M));
    CHECK(hipMemcpy(d_A, h_A, M, hipMemcpyHostToDevice));

    printf("\ncopy:\n");
    timing(d_A, d_B, N, 0);
    printf("\ntranspose with coalesced read:\n");
    timing(d_A, d_B, N, 1);
    printf("\ntranspose with shared coalesced read:\n");
    timing(d_A, d_B, N, 2);
    printf("\ntranspose with shared coalesced read and no bank conflicts:\n");
    timing(d_A, d_B, N, 3);

    CHECK(hipMemcpy(h_B, d_B, M, hipMemcpyDeviceToHost));
    if (N <= 10)
    {
        printf("A =\n");
        print_matrix(N, h_A);
        printf("\nB =\n");
        print_matrix(N, h_B);
    }

    free(h_A);
    free(h_B);
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    return 0;
}

void timing(const real *d_A, real *d_B, const int N, const int task)
{
    const dim3 block_size(TILE_DIM, TILE_DIM);  // size of matrix patch

    const int grid_size_x = (N + TILE_DIM - 1) / TILE_DIM;  // to make sure N columns are covered
    const int grid_size_y = grid_size_x;  // to make sure N rows are covered
    const dim3 grid_size(grid_size_x, grid_size_y);

    float t_sum = 0;
    float t2_sum = 0;
    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        switch (task)
        {
            case 0:
                copy<<<grid_size, block_size>>>(d_A, d_B, N);
                break;
            case 1:
                transpose1<<<grid_size, block_size>>>(d_A, d_B, N);
                break;
            case 2:
                transpose2<<<grid_size, block_size>>>(d_A, d_B, N);
                break;
            case 3:
                transpose3<<<grid_size, block_size>>>(d_A, d_B, N);
                break;
            default:
                printf("Error: wrong task\n");
                exit(1);
                break;
        }

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);
}

__global__ void copy(const real *A, real *B, const int N)
{
    const int nx = blockIdx.x * TILE_DIM + threadIdx.x;
    const int ny = blockIdx.y * TILE_DIM + threadIdx.y;
    const int index = ny * N + nx;
    if (nx < N && ny < N)
    {
        B[index] = A[index];
    }
}

__global__ void transpose1(const real *A, real *B, const int N)
{
    const int nx = blockIdx.x * blockDim.x + threadIdx.x;
    const int ny = blockIdx.y * blockDim.y + threadIdx.y;
    const int idx_in = ny * N + nx;
    const int idx_out = nx * N + ny;
    if (nx < N && ny < N)
    {
        B[idx_out] = A[idx_in];
    }
}

__global__ void transpose2(const real *A, real *B, const int N)
{
    __shared__ real S[TILE_DIM][TILE_DIM];

    const int bx = blockIdx.x * TILE_DIM;
    const int by = blockIdx.y * TILE_DIM;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int nx_in = bx + tx;
    const int ny_in = by + ty;
    const int idx_in = ny_in * N + nx_in;

    const int nx_out = by + tx;
    const int ny_out = bx + ty;
    const int idx_out = ny_out * N + nx_out;

    if (nx_in < N && ny_in < N)
    {
        S[ty][tx] = A[idx_in];
    }
    __syncthreads();
    
    if (nx_out < N && ny_out < N)
    {
        B[idx_out] = S[tx][ty];
    }       
}

__global__ void transpose3(const real *A, real *B, const int N)
{
    __shared__ real S[TILE_DIM][TILE_DIM + 1];

    const int bx = blockIdx.x * TILE_DIM;
    const int by = blockIdx.y * TILE_DIM;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int nx_in = bx + tx;
    const int ny_in = by + ty;
    const int idx_in = ny_in * N + nx_in;

    const int nx_out = by + tx;
    const int ny_out = bx + ty;
    const int idx_out = ny_out * N + nx_out;

    if (nx_in < N && ny_in < N)
    {
        S[ty][tx] = A[idx_in];
    }
    __syncthreads();
    
    if (nx_out < N && ny_out < N)
    {
        B[idx_out] = S[tx][ty];
    }      
}

void print_matrix(const int N, const real *A)
{
    for (int ny = 0; ny < N; ny++)
    {
        for (int nx = 0; nx < N; nx++)
        {
            printf("%g\t", A[ny * N + nx]);
        }
        printf("\n");
    }
}